#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <iostream>
#include <string>

#include "mandelbrot_gpu.cuh"

__device__
unsigned int mandelbrot_gpu_dev(SimpleComplexGPU value, int kDepth) {
    SimpleComplexGPU base = value;
    for (int i = 0; i < kDepth; ++i) {
        if (4 < value.magnitude_squared()) {
            return i;
        }
        value *= value;
        value += base;
    }
    return 0;
}

__global__
void mandelbrot_gpu(const double xmin, const  double xmax, const  double ymin, const  double ymax, int size_x, int size_y, unsigned int* image, int depth) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int nb_pixel = size_x * size_y;

    if (id >= nb_pixel) return;

    int proper_x = id % size_x;
    int proper_y = id / size_x;

    double delta_x = ((proper_x * 1.0) / size_x) * (xmax - xmin);
    double delta_y = ((proper_y * 1.0) / size_y) * (ymax - ymin);

    SimpleComplexGPU coordinate(xmin + delta_x, ymin + delta_y);
    unsigned int deviation = mandelbrot_gpu_dev(coordinate, depth);

    if (deviation > 0) {
        unsigned int r = ((deviation + 7) % 255) << 16;
        unsigned int g = ((deviation + 41) % 255) << 8;
        unsigned int b = (deviation + 127) % 255;
        image[id] = r | g | b;
    }
}

__host__
void render_gpu(double k_xmin,  double k_xmax,  double k_ymin,  double k_ymax, int size_x, int size_y, unsigned int* pixels, int depth) {
    unsigned int* d_image = 0;

    const int nb_pixel = size_x * size_y;

    hipMalloc((void**)&d_image, nb_pixel * sizeof(unsigned int));

    mandelbrot_gpu <<<(int)(nb_pixel/1024)+1, 1024>>>(k_xmin, k_xmax, k_ymin, k_ymax, size_x, size_y, d_image, depth);
    hipDeviceSynchronize();

    hipMemcpy(pixels, d_image, nb_pixel * sizeof(unsigned int), hipMemcpyKind::hipMemcpyDeviceToHost);
    hipFree(d_image);
}